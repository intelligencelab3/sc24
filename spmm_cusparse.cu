#include "spmm_cusparse.h"
#include "util.h"
#include <iostream>
using namespace std;

double spmm_cusparse(int *ptr, int *idx, float *val, float *vin, float *vout, int num_v, int num_e, int dim, int times)
{
    hipsparseHandle_t handle;
    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matB, matC;
    float alpha = 1.0;
    // float beta = 1.0;
    float beta = 0.0;
    float *buf = NULL;
    hipsparseCreate(&handle);
    if (ptr == NULL)
    {
        cout << "ptr is null !!!!" << endl;
    }
    hipsparseCreateCsr(&matA, num_v, num_v, num_e, ptr, idx, val, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
    hipsparseCreateDnMat(&matB, num_v, dim, dim, vin, HIP_R_32F, HIPSPARSE_ORDER_ROW);
    hipsparseCreateDnMat(&matC, num_v, dim, dim, vout, HIP_R_32F, HIPSPARSE_ORDER_ROW);
    size_t bufferSize = 0;
    hipsparseSpMM_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, matB, &beta, matC, HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize);
    hipMallocManaged(&buf, bufferSize);

    double ret = 0;
    if (times == 0)
    {
        hipsparseSpMM(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, matB, &beta, matC, HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT, buf);
        hipDeviceSynchronize();
    }
    else
    {
        times = 10;
        // warmup
        for (int i = 0; i < times; i++)
        {
            hipsparseSpMM(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, matB, &beta, matC, HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT, buf);
        }
        hipDeviceSynchronize();
        double measured_time = 0;
        for (int i = 0; i < times; i++)
        {
            timestamp(t0);
            hipsparseSpMM(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, matB, &beta, matC, HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT, buf);
            hipDeviceSynchronize();
            timestamp(t1);
            measured_time += getDuration(t0, t1);
        }
        ret = measured_time / times;
    }

    hipsparseDnMatGetValues(matC, (void **)&vout);
    hipsparseDestroy(handle);
    hipsparseDestroySpMat(matA);
    hipsparseDestroyDnMat(matB);
    hipsparseDestroyDnMat(matC);
    hipFree(buf);

    return ret;
}


double spmm_cusparse_coo(int *row, int *idx, float *val, float *vin, float *vout, int num_v, int num_e, int dim, int times)
{
    hipsparseHandle_t handle;
    
    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matB, matC;
    float alpha = 1.0;
    // float beta = 1.0;
    float beta = 0.0;
    float *buf = NULL;
    hipsparseCreate(&handle);
    
    hipsparseCreateCoo(&matA, num_v, num_v, num_e, row, idx, val, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
    hipsparseCreateDnMat(&matB, num_v, dim, dim, vin, HIP_R_32F, HIPSPARSE_ORDER_ROW);
    hipsparseCreateDnMat(&matC, num_v, dim, dim, vout, HIP_R_32F, HIPSPARSE_ORDER_ROW);
    size_t bufferSize = 0;
    hipsparseSpMM_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, matB, &beta, matC, HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize);
    hipMallocManaged(&buf, bufferSize);

    double ret = 0;
    if (times == 0)
    {
        hipsparseSpMM(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, matB, &beta, matC, HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT, buf);
        hipDeviceSynchronize();
    }
    else
    {
        times = 10;
        // warmup
        for (int i = 0; i < times; i++)
        {
            hipsparseSpMM(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, matB, &beta, matC, HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT, buf);
        }
        hipDeviceSynchronize();
        double measured_time = 0;
        for (int i = 0; i < times; i++)
        {
            timestamp(t0);
            hipsparseSpMM(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, matB, &beta, matC, HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT, buf);
            hipDeviceSynchronize();
            timestamp(t1);
            measured_time += getDuration(t0, t1);
        }
        ret = measured_time / times;
    }

    hipsparseDnMatGetValues(matC, (void **)&vout);
    hipsparseDestroy(handle);
    hipsparseDestroySpMat(matA);
    hipsparseDestroyDnMat(matB);
    hipsparseDestroyDnMat(matC);
    hipFree(buf);

    return ret;
}