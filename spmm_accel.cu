#include "hip/hip_runtime.h"
#include "spmm_accel.h"
#include "data.h"
#include <string>
#include <iostream>
#define CONSTINT const int

using namespace std;

extern string base_dir, graph;

string block_meta_dir = "../block_level_meta/";

const int DEG_BOUND = 12 * 32;
const int WARPS_PER_BLOCK = 12;

#define DIM_MUL(x) ((x + 31) / 32) * 32

__global__ void spmm_kernel_accel(const int *_block4, const int *coo_row, const int *idx, const float *val, const float *vin, float *vout, const int num_v, const int num_e, const int RHS_dim, const float *vout_ref)
{
    const int4 *block4 = reinterpret_cast<const int4 *>(_block4);
    const int4 b_info = block4[blockIdx.x]; //Consider use serveral int32 entries to store low-degree information to ease the bandwidth of bus

    CONSTINT block_degree_workload = b_info.x; 
    CONSTINT block_row_begin = b_info.y;
    CONSTINT block_loc_begin = b_info.z;
    CONSTINT block_info = b_info.w;

    CONSTINT block_degree=(block_degree_workload>>16);
    CONSTINT n_rows = (block_degree) <= DEG_BOUND ? block_info & 65535 : 1;//how many rows per block
    CONSTINT w_nz = block_degree <= DEG_BOUND ? block_info >> 16 : DEG_BOUND / WARPS_PER_BLOCK; //? put two info in 32 bit space each take 16bit how many nz's per warp
    CONSTINT row_nz = block_degree <= DEG_BOUND ? block_degree : block_info;// how many nz's per row
    CONSTINT workload = (block_degree_workload)&65535;
    extern __shared__ float out_cache[];
    // extern __shared__ float _val[];
    CONSTINT round_dim = DIM_MUL(RHS_dim); //what is RHS_dim? dimension of RHS matrix 
    // CONSTINT round_dim = RHS_dim;
    CONSTINT ext_1=round_dim/32;

    // printf("ext_1: %d \n", ext_1);
    // printf("RHS_dim: %d \n", RHS_dim);
    // printf("round_dim: %d \n", round_dim);
    // printf("(round_dim/ext_1): %d \n", (round_dim/ext_1));
    // printf("block_degree: %d, workload: %d\n", block_degree,workload);

    // return;

    CONSTINT warps_per_row = (row_nz + w_nz - 1) / w_nz;//?when RHS matrix is large

    #pragma unroll
    for (int ext = 0; ext < (RHS_dim + 31) / 32; ext++)//ext stands for column's dimension
    {        
        
        CONSTINT lane_id = (threadIdx.x +ext * blockDim.x)%round_dim;
 
        if(lane_id>=RHS_dim){

            return;

        }

         CONSTINT wid = (threadIdx.x + ext * blockDim.x) / round_dim;


        CONSTINT tid = wid * round_dim + lane_id;//tid== thread id


    if(block_degree<=12){
        if((wid%12)>=(n_rows)){
            // printf("wid: %d, how many rows(=warps) should be for this 1-deg chunk: %d\n",wid,n_rows);
            return;
        }
        // #pragma unroll
        // for (int jjj=0;jjj<ext_1;jjj++){


            if((block_degree==1)){
        

                    #pragma unroll
                    for(int jj=0;jj<(w_nz);jj++){   

                        const int nz_loc= block_loc_begin+(wid%12*32)-(wid%12)*(32-workload)+jj; // 1 warp to one nz's use for loop to access each of 30 nz's in a row, multiplied with 1 row of RHS(32 dim)
                        // printf("nz_loc: %d\n", nz_loc);
                        const int row_offset = (wid%12*32)-(wid%12)*(32-workload)+jj;


                        const float left_val = val[nz_loc];


                        float right_val = vin[idx[nz_loc] * RHS_dim+ lane_id];
                        vout[(block_row_begin + row_offset) * RHS_dim + lane_id] =left_val * right_val;

                        
                        
                    }
                    
        }

    else if((block_degree==2)){

            if((wid%12)>=(n_rows)){
                // printf("wid: %d, how many rows(=warps) should be for this 1-deg chunk: %d\n",wid,n_rows);
                return;
            }
                #pragma unroll
                 for(int jj=0;jj<(w_nz*2);jj+=2){   

                    const int nz_loc= block_loc_begin+(wid%12*32)-(wid%12)*(32-workload)+jj;
                    const int row_offset = ((wid%12*32)-(wid%12)*(32-workload)+jj)/2;
                    const float left_val_1 = __ldg(val+nz_loc);
                    const float left_val_2 = __ldg(val+nz_loc+1);


                    float right_val1 = vin[__ldg(idx + nz_loc)*RHS_dim + lane_id];
                    float right_val2 = vin[__ldg(idx + nz_loc+1)*RHS_dim + lane_id];

                    // for (int jjjj=0;jjjj<block_degree;jjjj++){
                    //     float left_val = __ldg(val+nz_loc+jjjj);

                    //     float right_val = vin[__ldg(idx + nz_loc+jjjj)*RHS_dim + lane_id];

                    //     vout[(block_row_begin + row_offset)*RHS_dim +lane_id] += right_val * left_val;
                    // }
                            
                    vout[(block_row_begin + row_offset)*RHS_dim +lane_id] +=left_val_1 * right_val1 + left_val_2 * right_val2;
                       
                }

            // }

            }

else if((block_degree==3)){

                    #pragma unroll
                    for(int jj=0;jj<(w_nz*3);jj+=3){   

                        const int nz_loc= block_loc_begin+(wid%12*32)-wid%12*(32-workload)+jj; // 1 warp to one nz's use for loop to access each of 30 nz's in a row, multiplied with 1 row of RHS(32 dim)
                        const int row_offset = ((wid%12*32)-wid%12*(32-workload)+jj)/3;
                        // const int cur_row= block_row_begin+((wid%12*32)-(wid%12)*(32%18)+jj)/3;

                        // const float left_val1 = __ldg(val+nz_loc);
                        // const float left_val2 = __ldg(val+nz_loc+1);
                        // const float left_val3 = __ldg(val+nz_loc+2);

                        // float right_val1 = vin[__ldg(idx + nz_loc)*RHS_dim + lane_id];
                        // float right_val2 = vin[__ldg(idx + nz_loc+1)*RHS_dim + lane_id];
                        // float right_val3 = vin[__ldg(idx + nz_loc+2)*RHS_dim + lane_id];

                        for (int jjjj=0;jjjj<block_degree;jjjj++){
                            float left_val = __ldg(val+nz_loc+jjjj);
    
                            float right_val = vin[__ldg(idx + nz_loc+jjjj)*RHS_dim + lane_id];
    
                            vout[(block_row_begin + row_offset)*RHS_dim +lane_id] += right_val * left_val;
                        }
 
                        // vout[(block_row_begin + row_offset)*RHS_dim + lane_id] +=left_val1 * right_val1 +left_val2 * right_val2+left_val3 * right_val3;
                                               

                                                    }

                    }


else if((block_degree==4)){

    #pragma unroll
  for(int jj=0;jj<(w_nz*4);jj+=4){   

    const int nz_loc= block_loc_begin+(wid%12*32)-(wid%12)*(32-workload)+jj; // 1 warp to one nz's use for loop to access each of 30 nz's in a row, multiplied with 1 row of RHS(32 dim)
    const int row_offset = ((wid%12*32)-(wid%12)*(32-workload)+jj)/4;
    // const int cur_row= block_row_begin+((wid%12*32)-(wid%12)*(32%24)+jj)/4;

    const float left_val1 = __ldg(val+nz_loc);
    const float left_val2 = __ldg(val+nz_loc+1);
    const float left_val3 = __ldg(val+nz_loc+2);
    const float left_val4 = __ldg(val+nz_loc+3);

    float right_val1 = vin[__ldg(idx + nz_loc)*RHS_dim + lane_id];
    float right_val2 = vin[__ldg(idx + nz_loc+1)*RHS_dim + lane_id];
    float right_val3 = vin[__ldg(idx + nz_loc+2)*RHS_dim + lane_id];
    float right_val4 = vin[__ldg(idx + nz_loc+3)*RHS_dim + lane_id];

    vout[(block_row_begin + row_offset)*RHS_dim +lane_id] +=left_val1 * right_val1 +left_val2 * right_val2+left_val3 * right_val3+left_val4 * right_val4;
                            
                                }
                }
    
    else if((block_degree==5)){

                #pragma unroll
              for(int jj=0;jj<(w_nz*5);jj+=5){   
            
                const int nz_loc= block_loc_begin+(wid%12*32)-(wid%12)*(32-workload)+jj; // 1 warp to one nz's use for loop to access each of 30 nz's in a row, multiplied with 1 row of RHS(32 dim)
                const int row_offset = ((wid%12*32)-(wid%12)*(32-workload)+jj)/5;
                // const int cur_row= block_row_begin+((wid%12*32)-(wid%12)*(32%25)+jj)/5;
            
                const float left_val1 = __ldg(val+nz_loc);
                const float left_val2 = __ldg(val+nz_loc+1);
                const float left_val3 = __ldg(val+nz_loc+2);
                const float left_val4 = __ldg(val+nz_loc+3);
                const float left_val5 = __ldg(val+nz_loc+4);
                

                float right_val1 = vin[__ldg(idx + nz_loc)*RHS_dim + lane_id];
                float right_val2 = vin[__ldg(idx + nz_loc+1)*RHS_dim + lane_id];
                float right_val3 = vin[__ldg(idx + nz_loc+2)*RHS_dim + lane_id];
                float right_val4 = vin[__ldg(idx + nz_loc+3)*RHS_dim + lane_id];
                float right_val5 = vin[__ldg(idx + nz_loc+4)*RHS_dim + lane_id];

                vout[(block_row_begin + row_offset)*RHS_dim + lane_id] +=left_val1 * right_val1 +left_val2 * right_val2+left_val3 * right_val3+left_val4 * right_val4 + left_val5 * right_val5;


                                }
                            } 
        else if((block_degree==6)){
        


                                        #pragma unroll
                                      for(int jj=0;jj<(w_nz*6);jj+=6){   
                                    
                                        const int nz_loc= block_loc_begin+(wid%12*32)-(wid%12)*(32-workload)+jj; // 1 warp to one nz's use for loop to access each of 30 nz's in a row, multiplied with 1 row of RHS(32 dim)
                                        const int row_offset = ((wid%12*32)-(wid%12)*(32-workload)+jj)/6;
                                        // const int cur_row= block_row_begin+((wid%12*32)-(wid%12)*(32%24)+jj)/6;
                                    
                                        const float left_val1 = __ldg(val+nz_loc);
                                        const float left_val2 = __ldg(val+nz_loc+1);
                                        const float left_val3 = __ldg(val+nz_loc+2);
                                        const float left_val4 = __ldg(val+nz_loc+3);
                                        const float left_val5 = __ldg(val+nz_loc+4);
                                        const float left_val6 = __ldg(val+nz_loc+5);
 
                                        float right_val1 = vin[__ldg(idx + nz_loc)*RHS_dim + lane_id];
                                        float right_val2 = vin[__ldg(idx + nz_loc+1)*RHS_dim + lane_id];
                                        float right_val3 = vin[__ldg(idx + nz_loc+2)*RHS_dim + lane_id];
                                        float right_val4 = vin[__ldg(idx + nz_loc+3)*RHS_dim + lane_id];
                                        float right_val5 = vin[__ldg(idx + nz_loc+4)*RHS_dim + lane_id];
                                        float right_val6 = vin[__ldg(idx + nz_loc+5)*RHS_dim + lane_id];
            
                                        vout[(block_row_begin + row_offset)*RHS_dim + lane_id] +=left_val1 * right_val1 +left_val2 * right_val2+left_val3 * right_val3+left_val4 * right_val4 + left_val5 * right_val5+ left_val6 * right_val6;

                                                                    }
                                                    } 

        else if((block_degree==7)){



                    #pragma unroll
                  for(int jj=0;jj<(w_nz*7);jj+=7){   
                
                    const int nz_loc= block_loc_begin+(wid%12*32)-(wid%12)*(32-workload)+jj; // 1 warp to one nz's use for loop to access each of 30 nz's in a row, multiplied with 1 row of RHS(32 dim)
                    const int row_offset = ((wid%12*32)-(wid%12)*(32-workload)+jj)/7;
                    // const int cur_row= block_row_begin+((wid%12*32)-(wid%12)*(32-workload)+jj)/7;
                
                    const float left_val1 = __ldg(val+nz_loc);
                    const float left_val2 = __ldg(val+nz_loc+1);
                    const float left_val3 = __ldg(val+nz_loc+2);
                    const float left_val4 = __ldg(val+nz_loc+3);
                    const float left_val5 = __ldg(val+nz_loc+4);
                    const float left_val6 = __ldg(val+nz_loc+5);
                    const float left_val7 = __ldg(val+nz_loc+6);

                // for(int jjj=0;jjj<ext_1;jjj++){
                    float right_val1 = vin[__ldg(idx + nz_loc)*RHS_dim + lane_id];
                    float right_val2 = vin[__ldg(idx + nz_loc+1)*RHS_dim + lane_id];
                    float right_val3 = vin[__ldg(idx + nz_loc+2)*RHS_dim + lane_id];
                    float right_val4 = vin[__ldg(idx + nz_loc+3)*RHS_dim + lane_id];
                    float right_val5 = vin[__ldg(idx + nz_loc+4)*RHS_dim + lane_id];
                    float right_val6 = vin[__ldg(idx + nz_loc+5)*RHS_dim + lane_id];
                    float right_val7 = vin[__ldg(idx + nz_loc+6)*RHS_dim + lane_id];


                    vout[(block_row_begin + row_offset)*RHS_dim +lane_id] +=left_val1 * right_val1 +left_val2 * right_val2+left_val3 * right_val3+left_val4 * right_val4 + left_val5 * right_val5+ left_val6 * right_val6 +left_val7 * right_val7;
                

                                                }
                                } 

        else if((block_degree==8)){
               
                                        #pragma unroll
                                      for(int jj=0;jj<(w_nz*8);jj+=8){   
                                    
                                        const int nz_loc= block_loc_begin+(wid%12*32)-(wid%12)*(32-workload)+jj; // 1 warp to one nz's use for loop to access each of 30 nz's in a row, multiplied with 1 row of RHS(32 dim)
                                        const int row_offset = ((wid%12*32)-(wid%12)*(32-workload)+jj)/8;
                                        // const int cur_row= block_row_begin+((wid%12*32)-(wid%12)*(32-workload)+jj)/8;
                                    
                                        const float left_val1 = __ldg(val+nz_loc);
                                        const float left_val2 = __ldg(val+nz_loc+1);
                                        const float left_val3 = __ldg(val+nz_loc+2);
                                        const float left_val4 = __ldg(val+nz_loc+3);
                                        const float left_val5 = __ldg(val+nz_loc+4);
                                        const float left_val6 = __ldg(val+nz_loc+5);
                                        const float left_val7 = __ldg(val+nz_loc+6);
                                        const float left_val8 = __ldg(val+nz_loc+7);
                    

            

                            float right_val1 = vin[__ldg(idx + nz_loc)*RHS_dim + lane_id];
                            float right_val2 = vin[__ldg(idx + nz_loc+1)*RHS_dim + lane_id];
                            float right_val3 = vin[__ldg(idx + nz_loc+2)*RHS_dim + lane_id];
                            float right_val4 = vin[__ldg(idx + nz_loc+3)*RHS_dim + lane_id];
                            float right_val5 = vin[__ldg(idx + nz_loc+4)*RHS_dim + lane_id];
                            float right_val6 = vin[__ldg(idx + nz_loc+5)*RHS_dim + lane_id];
                            float right_val7 = vin[__ldg(idx + nz_loc+6)*RHS_dim + lane_id];
                            float right_val8 = vin[__ldg(idx + nz_loc+7)*RHS_dim + lane_id];

                        
                            vout[(block_row_begin + row_offset)*RHS_dim +lane_id] +=left_val1 * right_val1 +left_val2 * right_val2+left_val3 * right_val3+left_val4 * right_val4 + left_val5 * right_val5+ left_val6 * right_val6 +left_val7 * right_val7+left_val8 * right_val8;
                        


                                                                    }
                                                    } 
                else if((block_degree==9)){
                    #pragma unroll
                    for(int jj=0;jj<(w_nz*9);jj+=9){   
                  
                      const int nz_loc= block_loc_begin+(wid%12*32)-(wid%12)*(32-workload)+jj; // 1 warp to one nz's use for loop to access each of 30 nz's in a row, multiplied with 1 row of RHS(32 dim)
                      const int row_offset = ((wid%12*32)-(wid%12)*(32-workload)+jj)/9;
                      // const int cur_row= block_row_begin+((wid%12*32)-(wid%12)*(32-workload)+jj)/8;
                  
                      const float left_val1 = __ldg(val+nz_loc);
                      const float left_val2 = __ldg(val+nz_loc+1);
                      const float left_val3 = __ldg(val+nz_loc+2);
                      const float left_val4 = __ldg(val+nz_loc+3);
                      const float left_val5 = __ldg(val+nz_loc+4);
                      const float left_val6 = __ldg(val+nz_loc+5);
                      const float left_val7 = __ldg(val+nz_loc+6);
                      const float left_val8 = __ldg(val+nz_loc+7);
                      const float left_val9 = __ldg(val+nz_loc+8);
  



                        float right_val1 = vin[__ldg(idx + nz_loc)*RHS_dim + lane_id];
                        float right_val2 = vin[__ldg(idx + nz_loc+1)*RHS_dim + lane_id];
                        float right_val3 = vin[__ldg(idx + nz_loc+2)*RHS_dim + lane_id];
                        float right_val4 = vin[__ldg(idx + nz_loc+3)*RHS_dim + lane_id];
                        float right_val5 = vin[__ldg(idx + nz_loc+4)*RHS_dim + lane_id];
                        float right_val6 = vin[__ldg(idx + nz_loc+5)*RHS_dim + lane_id];
                        float right_val7 = vin[__ldg(idx + nz_loc+6)*RHS_dim + lane_id];
                        float right_val8 = vin[__ldg(idx + nz_loc+7)*RHS_dim + lane_id];
                        float right_val9 = vin[__ldg(idx + nz_loc+8)*RHS_dim + lane_id];

      
          vout[(block_row_begin + row_offset)*RHS_dim +lane_id] +=left_val1 * right_val1 +left_val2 * right_val2+left_val3 * right_val3+left_val4 * right_val4 + left_val5 * right_val5+ left_val6 * right_val6 +left_val7 * right_val7+left_val8 * right_val8+left_val9 * right_val9;
      

                }
                                                }

                else if((block_degree==10)){
                    #pragma unroll
                    for(int jj=0;jj<(w_nz*10);jj+=10){   
                  
                      const int nz_loc= block_loc_begin+(wid%12*32)-(wid%12)*(32-workload)+jj; // 1 warp to one nz's use for loop to access each of 30 nz's in a row, multiplied with 1 row of RHS(32 dim)
                      const int row_offset = ((wid%12*32)-(wid%12)*(32-workload)+jj)/10;
                      // const int cur_row= block_row_begin+((wid%12*32)-(wid%12)*(32-workload)+jj)/8;
                  
                      const float left_val1 = __ldg(val+nz_loc);
                      const float left_val2 = __ldg(val+nz_loc+1);
                      const float left_val3 = __ldg(val+nz_loc+2);
                      const float left_val4 = __ldg(val+nz_loc+3);
                      const float left_val5 = __ldg(val+nz_loc+4);
                      const float left_val6 = __ldg(val+nz_loc+5);
                      const float left_val7 = __ldg(val+nz_loc+6);
                      const float left_val8 = __ldg(val+nz_loc+7);
                      const float left_val9 = __ldg(val+nz_loc+8);
                      const float left_val10 = __ldg(val+nz_loc+9);
  



                        float right_val1 = vin[__ldg(idx + nz_loc)*RHS_dim + lane_id];
                        float right_val2 = vin[__ldg(idx + nz_loc+1)*RHS_dim + lane_id];
                        float right_val3 = vin[__ldg(idx + nz_loc+2)*RHS_dim + lane_id];
                        float right_val4 = vin[__ldg(idx + nz_loc+3)*RHS_dim + lane_id];
                        float right_val5 = vin[__ldg(idx + nz_loc+4)*RHS_dim + lane_id];
                        float right_val6 = vin[__ldg(idx + nz_loc+5)*RHS_dim + lane_id];
                        float right_val7 = vin[__ldg(idx + nz_loc+6)*RHS_dim + lane_id];
                        float right_val8 = vin[__ldg(idx + nz_loc+7)*RHS_dim + lane_id];
                        float right_val9 = vin[__ldg(idx + nz_loc+8)*RHS_dim + lane_id];
                        float right_val10 = vin[__ldg(idx + nz_loc+9)*RHS_dim + lane_id];

      
          vout[(block_row_begin + row_offset)*RHS_dim +lane_id] +=left_val1 * right_val1 +left_val2 * right_val2+left_val3 * right_val3+left_val4 * right_val4 + left_val5 * right_val5+ left_val6 * right_val6 +left_val7 * right_val7+left_val8 * right_val8+left_val9 * right_val9+left_val10 * right_val10;
      

                }                                 
                                                    
            }

            else if((block_degree==11)){
                #pragma unroll
                for(int jj=0;jj<(w_nz*11);jj+=11){   
              
                  const int nz_loc= block_loc_begin+(wid%12*32)-(wid%12)*(32-workload)+jj; // 1 warp to one nz's use for loop to access each of 30 nz's in a row, multiplied with 1 row of RHS(32 dim)
                  const int row_offset = ((wid%12*32)-(wid%12)*(32-workload)+jj)/11;
                  // const int cur_row= block_row_begin+((wid%12*32)-(wid%12)*(32-workload)+jj)/8;
              
                  const float left_val1 = __ldg(val+nz_loc);
                  const float left_val2 = __ldg(val+nz_loc+1);
                  const float left_val3 = __ldg(val+nz_loc+2);
                  const float left_val4 = __ldg(val+nz_loc+3);
                  const float left_val5 = __ldg(val+nz_loc+4);
                  const float left_val6 = __ldg(val+nz_loc+5);
                  const float left_val7 = __ldg(val+nz_loc+6);
                  const float left_val8 = __ldg(val+nz_loc+7);
                  const float left_val9 = __ldg(val+nz_loc+8);
                  const float left_val10 = __ldg(val+nz_loc+9);
                  const float left_val11 = __ldg(val+nz_loc+10);




                    float right_val1 = vin[__ldg(idx + nz_loc)*RHS_dim + lane_id];
                    float right_val2 = vin[__ldg(idx + nz_loc+1)*RHS_dim + lane_id];
                    float right_val3 = vin[__ldg(idx + nz_loc+2)*RHS_dim + lane_id];
                    float right_val4 = vin[__ldg(idx + nz_loc+3)*RHS_dim + lane_id];
                    float right_val5 = vin[__ldg(idx + nz_loc+4)*RHS_dim + lane_id];
                    float right_val6 = vin[__ldg(idx + nz_loc+5)*RHS_dim + lane_id];
                    float right_val7 = vin[__ldg(idx + nz_loc+6)*RHS_dim + lane_id];
                    float right_val8 = vin[__ldg(idx + nz_loc+7)*RHS_dim + lane_id];
                    float right_val9 = vin[__ldg(idx + nz_loc+8)*RHS_dim + lane_id];
                    float right_val10 = vin[__ldg(idx + nz_loc+9)*RHS_dim + lane_id];
                    float right_val11 = vin[__ldg(idx + nz_loc+10)*RHS_dim + lane_id];

  
      vout[(block_row_begin + row_offset)*RHS_dim +lane_id] +=left_val1 * right_val1 +left_val2 * right_val2+left_val3 * right_val3+left_val4 * right_val4 + left_val5 * right_val5+ left_val6 * right_val6 +left_val7 * right_val7+left_val8 * right_val8+left_val9 * right_val9+left_val10 * right_val10+left_val11 * right_val11;
  
            }

        }
        else if((block_degree==12)){
            #pragma unroll
            for(int jj=0;jj<(w_nz*12);jj+=12){   
          
              const int nz_loc= block_loc_begin+(wid%12*32)-(wid%12)*(32-workload)+jj; // 1 warp to one nz's use for loop to access each of 30 nz's in a row, multiplied with 1 row of RHS(32 dim)
              const int row_offset = ((wid%12*32)-(wid%12)*(32-workload)+jj)/12;
              // const int cur_row= block_row_begin+((wid%12*32)-(wid%12)*(32-workload)+jj)/8;
          
              const float left_val1 = __ldg(val+nz_loc);
              const float left_val2 = __ldg(val+nz_loc+1);
              const float left_val3 = __ldg(val+nz_loc+2);
              const float left_val4 = __ldg(val+nz_loc+3);
              const float left_val5 = __ldg(val+nz_loc+4);
              const float left_val6 = __ldg(val+nz_loc+5);
              const float left_val7 = __ldg(val+nz_loc+6);
              const float left_val8 = __ldg(val+nz_loc+7);
              const float left_val9 = __ldg(val+nz_loc+8);
              const float left_val10 = __ldg(val+nz_loc+9);
              const float left_val11 = __ldg(val+nz_loc+10);
              const float left_val12 = __ldg(val+nz_loc+11);


                float right_val1 = vin[__ldg(idx + nz_loc)*RHS_dim + lane_id];
                float right_val2 = vin[__ldg(idx + nz_loc+1)*RHS_dim + lane_id];
                float right_val3 = vin[__ldg(idx + nz_loc+2)*RHS_dim + lane_id];
                float right_val4 = vin[__ldg(idx + nz_loc+3)*RHS_dim + lane_id];
                float right_val5 = vin[__ldg(idx + nz_loc+4)*RHS_dim + lane_id];
                float right_val6 = vin[__ldg(idx + nz_loc+5)*RHS_dim + lane_id];
                float right_val7 = vin[__ldg(idx + nz_loc+6)*RHS_dim + lane_id];
                float right_val8 = vin[__ldg(idx + nz_loc+7)*RHS_dim + lane_id];
                float right_val9 = vin[__ldg(idx + nz_loc+8)*RHS_dim + lane_id];
                float right_val10 = vin[__ldg(idx + nz_loc+9)*RHS_dim + lane_id];
                float right_val11 = vin[__ldg(idx + nz_loc+10)*RHS_dim + lane_id];
                float right_val12 = vin[__ldg(idx + nz_loc+11)*RHS_dim + lane_id];


  vout[(block_row_begin + row_offset)*RHS_dim +lane_id] +=left_val1 * right_val1 +left_val2 * right_val2+left_val3 * right_val3+left_val4 * right_val4 + left_val5 * right_val5+ left_val6 * right_val6 +left_val7 * right_val7+left_val8 * right_val8+left_val9 * right_val9+left_val10 * right_val10+left_val11 * right_val11+left_val12 * right_val12;

        }
        }

    // }
    }
            else if(block_degree>12){


                    
                    CONSTINT warp_loc_row = wid / warps_per_row; 
                    CONSTINT warp_loc_col = wid % warps_per_row * w_nz;
            
                    if (warp_loc_row >= n_rows)
                    {
                        return;
                    }
            
                    //decide how many nz's for 1 warp
                    //based on the degree of incoming row
                    //find nz's location at one time
                    //then perform the spmm mul_add
            
            #pragma unroll
                    for (int i = 0; i < w_nz; i++)
                    {
                        if (i + warp_loc_col >= row_nz)
                        {
                            break;
                        }
                        if (i == 0)
                        {
                            out_cache[tid] = 0;

                        }
                        const int nz_loc = block_loc_begin + warp_loc_row * row_nz + i + warp_loc_col;
                        const float left_val = __ldg(val + nz_loc);
            
                        float right_val = vin[__ldg(idx + nz_loc) * RHS_dim + lane_id];
                        out_cache[tid] += left_val * right_val;

                    }
            
                    // atomicAdd(&vout[(block_row_begin + warp_loc_row) * RHS_dim + lane_id], out_cache[wid * round_dim + lane_id]);
                    if (warps_per_row > 1)
                    {
                        atomicAdd(&vout[(block_row_begin + warp_loc_row) * RHS_dim + lane_id], out_cache[tid]);
                        
                    }
                    else
                    {
                        if (block_degree <= DEG_BOUND)
                        {
            
                            vout[(block_row_begin + wid) * RHS_dim + lane_id] = out_cache[tid];
                        }
            
                        else
                        {
                            atomicAdd(&vout[(block_row_begin + wid) * RHS_dim + lane_id], out_cache[tid]);
                        }
                    }
                    
                
            }
        }

 
        }


    
   
                    

            
    




void SPMM_ACCEL::run(int dim)
{    
    int shared_size = (WARPS_PER_BLOCK + 0 * WARPS_PER_BLOCK / 2) * DIM_MUL(dim) * sizeof(float);//is this dim the RHS_dim?
    spmm_kernel_accel<<<grid, block, shared_size>>>(_block4, 0, idx, val, vin, vout, num_v, num_e, dim, 0);
}

double SPMM_ACCEL::do_test(bool timing, int dim)
{
    // hipMallocManaged(&coo_row, num_e * sizeof(int));
    // int k = 0;
    // for (int i = 0; i < num_v; i++)
    // {
    //     for (int j = 0; j < ptr[i + 1] - ptr[i]; j++)
    //     {
    //         coo_row[k++] = i;
    //     }
    // }

    // int block_num = cuda_read_array(&this->_block4, "../block_level_meta/" + this->_graph + ".block4") / 4;
    int block_num = cuda_read_array(&this->_block4, block_meta_dir + this->_graph + ".block4") / 4;
 
    if (!timing)
    {
    //    cout << "block num = " << block_num << endl;
    }

    grid.x = block_num;

    // printf("block_num:%d\n",block_num);

    // block.x = DIM_MUL(dim);
    // block.y = WARPS_PER_BLOCK;
    block.x = WARPS_PER_BLOCK * 32;

    double ret = timing_body(timing, dim); //probably too few blocks

    // hipFree(coo_row);
    hipFree(this->_block4);
    return ret;
}